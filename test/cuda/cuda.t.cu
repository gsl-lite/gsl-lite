#include "hip/hip_runtime.h"
//
// gsl-lite is based on GSL: Guidelines Support Library.
// For more information see https://github.com/martinmoene/gsl-lite
//
// Copyright (c) 2015 Martin Moene
// Copyright (c) 2015 Microsoft Corporation. All rights reserved. 
// 
// This code is licensed under the MIT License (MIT). 
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, 
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN 
// THE SOFTWARE. 

#include "../gsl-lite.t.hpp"

#include <memory>
#include <sstream>


// CUDA error checking macro borrowed from https://stackoverflow.com/a/14038590 and adapted.
#define myCudaErrchk(ans) ( myCudaErrchkImpl( (ans), #ans, __FILE__, __LINE__ ) )
inline void myCudaErrchkImpl( hipError_t code, char const * expr, char const * file, int line )
{
    if ( code != hipSuccess )
    {
        std::ostringstream sstr;
        sstr << "CUDA error in '" << expr << "' at " << file << "(" << line << "): "
             << hipGetErrorString( code );
        throw std::runtime_error( sstr.str() );
   }
}


struct CudaMallocDeleter
{
    template< class T >
    void operator ()( T* data ) const
    {
        hipFree( data );
    }
};

template< class T >
using CudaUniquePtr = std::unique_ptr<T, CudaMallocDeleter>;

template< class T >
struct CudaAllocHelper
{
    static CudaUniquePtr<T> cudaAllocUnique()
    {
        T* ptr;
        auto ec = hipMalloc( &ptr, sizeof( T ) );
        if ( ec != hipSuccess ) throw std::bad_alloc{ };
        return CudaUniquePtr<T>( ptr );
    }
};
template< class T >
struct CudaAllocHelper<T[]>
{
    static CudaUniquePtr<T[]> cudaAllocUnique( std::size_t n )
    {
        T* ptr;
        auto ec = hipMalloc( &ptr, sizeof( T ) * n );
        if ( ec != hipSuccess ) throw std::bad_alloc{ };
        return CudaUniquePtr<T[]>( ptr );
    }
};
template< class T >
CudaUniquePtr<T> cudaAllocUnique()
{
    return CudaAllocHelper<T>::cudaAllocUnique();
}
template< class T >
CudaUniquePtr<T> cudaAllocUnique( std::size_t n )
{
    return CudaAllocHelper<T>::cudaAllocUnique(n);
}


__global__ void preconditionAssertionKernel( int i, int j )
{
    gsl_Expects( i >= 0 );
    gsl_ExpectsAudit( i < j );
    gsl_Ensures( i >= 0 );
    gsl_EnsuresAudit( i < j );
    gsl_Assert( i >= 0 );
    gsl_AssertAudit( i < j );
}

CASE( "CUDA: Precondition/postcondition checks and assertions can be used in kernel code" )
{
    preconditionAssertionKernel<<<1, 1>>>( 0, 1 );
    myCudaErrchk( hipPeekAtLastError() );  // check for invalid launch arguments
    myCudaErrchk( hipDeviceSynchronize() );  // check for execution errors
}


__global__ void spanKernel( gsl::span< int > span )
{
    int* data = span.data();
    gsl_CONFIG_SPAN_INDEX_TYPE size = span.size();
    if (size > 0)
    {
        span[0] = 42;
        at( span, 0 ) = 42;
    }
}

CASE( "CUDA: span<> can be passed to kernel code" )
{
    spanKernel<<<1, 1>>>( gsl::span< int >( ) );
    myCudaErrchk( hipPeekAtLastError() );  // check for invalid launch arguments
    myCudaErrchk( hipDeviceSynchronize() );  // check for execution errors
}

CASE( "CUDA: span<> can be used in kernel code" )
{
    std::size_t n = 3;
    auto array = cudaAllocUnique<int[]>(n);
    auto span = gsl::make_span(array.get(), n);
    spanKernel<<<1, 1>>>( span );
    myCudaErrchk( hipPeekAtLastError() );  // check for invalid launch arguments
    myCudaErrchk( hipDeviceSynchronize() );  // check for execution errors
}


__global__ void notNullRawKernel( gsl::not_null< int* > ptr )
{
    *ptr = 1;
    auto ptr2 = ptr;
    *ptr2 = 2;
    ptr = ptr2;
}

// Not supported yet because `std::unique_ptr<>` member functions are neither `constexpr` nor `__host__ __device__`.
//__global__ void notNullUniqueKernel( gsl::not_null< CudaUniquePtr< int > > ptr )
//{
//    *ptr = 3;
//    auto ptr2 = std::move( ptr );
//    *ptr2 = 4;
//    ptr = std::move( ptr2 );
//}

CASE( "CUDA: not_null<> can be passed to and used in kernel code" )
{
    auto pi = cudaAllocUnique<int>();

    notNullRawKernel<<<1, 1>>>( gsl::make_not_null( pi.get() ) );
    myCudaErrchk( hipPeekAtLastError() );  // check for invalid launch arguments
    myCudaErrchk( hipDeviceSynchronize() );  // check for execution errors

    //notNullUniqueKernel<<<1, 1>>>( gsl::make_not_null( std::move( pi ) ) );
    //myCudaErrchk( hipPeekAtLastError() );  // check for invalid launch arguments
    //myCudaErrchk( hipDeviceSynchronize() );  // check for execution errors
}


__global__ void failFastKernel()
{
    gsl_FailFast();
}

CASE( "CUDA: gsl_FailFast() can be used in kernel code" )
{
    failFastKernel<<<1, 1>>>();
    myCudaErrchk( hipPeekAtLastError() );
    EXPECT( hipDeviceSynchronize() != hipSuccess );
    // Note that executing a trap instruction or an assertion in a CUDA kernel destroys the context, so this should be
    // the last test case we execute.
}

