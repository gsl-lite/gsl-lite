#include "hip/hip_runtime.h"
//
// gsl-lite is based on GSL: Guidelines Support Library.
// For more information see https://github.com/martinmoene/gsl-lite
//
// Copyright (c) 2015 Martin Moene
// Copyright (c) 2015 Microsoft Corporation. All rights reserved. 
// 
// This code is licensed under the MIT License (MIT). 
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, 
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN 
// THE SOFTWARE. 

#include "../gsl-lite.t.hpp"

#include <memory>


__global__ void preconditionAssertionKernel( int i, int j )
{
    gsl_Expects( i >= 0 );
    gsl_ExpectsAudit( i < j );
}

CASE( "CUDA: Precondition and postcondition assertions can be used in kernel code" )
{
    preconditionAssertionKernel<<<1, 1>>>( 0, 1 );
    // TODO: check for failure
}


__global__ void spanKernel( gsl::span< int > span )
{
    int* data = span.data();
    gsl_CONFIG_SPAN_INDEX_TYPE size = span.size();
    if (size > 0)
    {
        span[ 0 ] = 42;
        at( span, 0 ) = 42;
    }
    // TODO: add more tests
}

CASE( "CUDA: span<> can be used in kernel code" )
{
    spanKernel<<<1, 1>>>( gsl::span< int >( ) );
    // TODO: check for failure
    // TODO: add more tests
}


__global__ void notNullKernel( gsl::not_null< int* > ptr )
{
    // TODO: add more tests
}

CASE( "CUDA: not_null<> can be used in kernel code" )
{
    // TODO: run kernel
    // TODO: check for failure
    // TODO: add more tests
}
